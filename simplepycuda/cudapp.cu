#include "cudapp.h"

extern "C" int cudappGetDeviceCount() {
	int deviceCount;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	return deviceCount;
}

extern "C" void cudappSetDevice(int index) {
	checkCudaErrors(hipSetDevice(index));
}

extern "C" void cudappDeviceSynchronize() {
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" void cudappDeviceReset() {
	checkCudaErrors(hipDeviceReset());
}

extern "C" void* cudappMalloc(size_t nbytes) {
	void* p;
	checkCudaErrors(hipMalloc(&p, nbytes));
	return p;
}

extern "C" void cudappFree(void* p) {
	checkCudaErrors(hipFree(p));
}

extern "C" void cudappMemset(void* p, unsigned char v, size_t count) {
	checkCudaErrors(hipMemset(p, v, count));
}

extern "C" void cudappMemcpyHostToDevice(void* d, void* h, size_t nbytes) {
	checkCudaErrors(hipMemcpy(d, h, nbytes, hipMemcpyHostToDevice));
}

extern "C" void cudappMemcpyDeviceToHost(void* h, void* d, size_t nbytes) {
	checkCudaErrors(hipMemcpy(h, d, nbytes, hipMemcpyDeviceToHost));
}

// event handling (timer)

extern "C" void* cudappEventCreate() {
	hipEvent_t* ev = new hipEvent_t;
	checkCudaErrors(hipEventCreate(ev));
	return ev;
}

extern "C" void cudappEventRecord(void* event, size_t stream) {
	hipEvent_t* ev = (hipEvent_t*) event;
	checkCudaErrors(hipEventRecord(*ev, (hipStream_t) stream));
}

extern "C" void cudappEventSynchronize(void* event) {
	hipEvent_t* ev = (hipEvent_t*) event;
	checkCudaErrors(hipEventSynchronize(*ev));
}

extern "C" float cudappEventElapsedTime(void* event1, void* event2) {
	hipEvent_t* ev1 = (hipEvent_t*) event1;
	hipEvent_t* ev2 = (hipEvent_t*) event2;
	float f;
	checkCudaErrors(hipEventElapsedTime(&f, *ev1, *ev2));
	return f;
}

extern "C" void cudappEventDestroy(void* event) {
	hipEvent_t* ev = (hipEvent_t*) event;
	checkCudaErrors(hipEventDestroy(*ev));
	delete ev;
}

extern "C" hipError_t cudappGetLastError() {
	return hipGetLastError();
}
